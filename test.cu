
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>

// Kernel function to add the elements of two arrays
__global__
void add(long n, float *x, float *y)
{
  long index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < n)
    y[index] = x[index] + y[index];
}

int main(void)
{


  long N = 1<<30;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipEvent_t start, stop;
  float elapsedTime;
  for(int i=0; i<10; ++i){
    hipEventCreate(&start);
    hipEventRecord(start,0);

    // Run kernel on 1M elements on the GPU
    int blockSize = 1024;
    long numBlocks = (N + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(N, x, y);

    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start,stop);
    printf("Elapsed time : %f ms\n" ,elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    // float maxError = 0.0f;
    // for (int i = 0; i < N; i++)
    //   maxError = fmax(maxError, fabs(y[i]-3.0f));
    // std::cout << "Max error: " << maxError << std::endl;

  }
  // Free memory
  hipFree(x);
  hipFree(y);


  
  return 0;
}