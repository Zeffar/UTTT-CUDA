#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int deviceId;
    hipGetDevice(&deviceId); // Get the current device ID

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, deviceId); // Get the device properties

    std::cout << "Max threads per multiprocessor: " << props.maxThreadsPerMultiProcessor << "\n";
    std::cout << "Number of multiprocessors: " << props.multiProcessorCount << "\n";
    std::cout << "Max threads per block: " << props.maxThreadsPerBlock << "\n";

    int maxThreads = props.maxThreadsPerMultiProcessor * props.multiProcessorCount;
std::cout << "Maximum threads that can be run concurrently: " << maxThreads << std::endl;
return 0;
}